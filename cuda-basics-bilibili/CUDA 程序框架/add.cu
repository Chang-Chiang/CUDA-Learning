// nvcc -o add add.cu -arch=compute_60 -code=sm_60

#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>

const double EPSILON = 1.0e-15;
const double a = 1.23;
const double b = 2.34;
const double c = 3.57;

void __global__ add(const double *x, const double *y, double *z);

void check(const double *z, const int N);

int main(void)
{
    const int N = 100000000;
    const int M = sizeof(double) * N;

    // 主机分配内存
    double *h_x = (double*) malloc(M);
    double *h_y = (double*) malloc(M);
    double *h_z = (double*) malloc(M);

    // 初始化数组
    for (int n = 0; n < N; ++n)
    {
        h_x[n] = a;
        h_y[n] = b;
    }

    // 设备分配内存, GPU 显存
    double *d_x, *d_y, *d_z;
    hipMalloc((void **)&d_x, M);
    hipMalloc((void **)&d_y, M);
    hipMalloc((void **)&d_z, M);

    hipMemcpy(d_x, h_x, M, hipMemcpyHostToDevice);
    hipMemcpy(d_y, h_y, M, hipMemcpyHostToDevice);

    const int block_size = 128;
    const int grid_size = N / block_size;
    add<<<grid_size, block_size>>>(d_x, d_y, d_z);

    hipMemcpy(h_z, d_z, M, hipMemcpyDeviceToHost);
    check(h_z, N);

    free(h_x);
    free(h_y);
    free(h_z);

    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_z);

    return 0;
}

/**
 * @brief 核函数, GPU 上运行
 *
 * @param x 数组 x 的地址
 * @param y 数组 y 的地址
 * @param z 数组 z 的地址
 */
void __global__ add(const double *x, const double *y, double *z)
{
    const int n = blockDim.x * blockIdx.x + threadIdx.x;
    z[n] = x[n] + y[n];
}

/**
 * @brief 校验数组 z 每一个元素值是否等于 c
 *
 * @param z 数组 z 的地址
 * @param N 数组 z 的长度
 */
void check(const double *z, const int N)
{
    bool has_error = false;
    for (int n = 0; n < N; ++n)
    {
        if (fabs(z[n] - c) > EPSILON)
        {
            has_error = true;
        }
    }
    printf("%s\n", has_error ? "Has errors" : "No errors");
}

